#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void saxpy(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        y[i] = a * x[i] + y[i];
}

void benchmark_saxpy(int n) {
    float *x, *y, *d_x, *d_y;
    float a = 2.0f;

    size_t size = n * sizeof(float);
    x = (float*)malloc(size);
    y = (float*)malloc(size);

    for (int i = 0; i < n; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    hipEventRecord(start);
    saxpy<<<blocksPerGrid, threadsPerBlock>>>(n, a, d_x, d_y);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("N = %8d | Time = %8.4f ms\n", n, milliseconds);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
}

int main() {
    printf("SAXPY Benchmark (GPU only kernel time)\n");
    for (int exp = 15; exp <= 25; exp++) {
        int n = 1 << exp;
        benchmark_saxpy(n);
    }
    return 0;
}
